#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM_THREADS 16
#define min(a, b) (((a) < (b)) ? (a) : (b))
#define row_major(i, j, num_rows) ((i) * (num_rows) + (j))

__global__ void dw_conv(double *X, double *F_DW, double *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int stride_h, int stride_w) {
	
	// Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x; 
    int c = threadIdx.y + blockIdx.y * blockDim.y; 

    // Pre-computations
    int mat_size = W_in * H_in;
    int f_size = W_f * H_f;
    int img_size = mat_size * C_in;

    int temp_out_img_size = W_out * H_out;
    int temp_out_size = temp_out_img_size * N_dw * C_in;

    // PTRS TO IMG IN BATCH
    double *curr_img = X + b * img_size;
    double *curr_out = O + b * temp_out_size;

    // Do 2D Convolution channelwise
    double *curr_channel = curr_img + mat_size * c;

    // Filters are 2D
    for (int f = 0; f < N_dw; f += 1)
    {
        for (int w = 0; w < W_out; w += 1)
        {
            for (int h = 0; h < H_out; h += 1)
            {
                // MICROKERNEL - tile if needed.
                for (int w_f = 0; w_f < W_f; w_f += 1)
                {
                    for (int h_f = 0; h_f < H_f; h_f += 1)
                    {
                        // PTR TO CURRENT POSITION IN FILTER
                        double *f_curr = F_DW + f_size * (c * N_dw + f) + row_major(h_f, w_f, W_f);

                        // PTR TO INPUT POSITION
                        int h_curr = h_f + stride_h * h;
                        int w_curr = w_f + stride_w * w;
                        double *curr_inp = curr_channel + row_major(h_curr, w_curr, W_in);

                        // PTR TO INPUT POSITION
                        double *curr_out_xy = curr_out + temp_out_img_size * (c * N_dw + f) + row_major(h, w, W_out);

                        // CONVOLVE
                        *curr_out_xy = *curr_out_xy + *f_curr * *curr_inp;
                    }
                }
            }
        }
    }
}

__global__ void pw_conv(double *X, double *F_1D, double *O, int B, int H_in, int W_in, int C_in, int C_out)
{
    // Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x; 

    int mat_size = W_in * H_in;
    int img_size = mat_size * C_in;
    int out_size = mat_size * C_out;

    double *curr_img = X + b * img_size;
    double *curr_out = O + b * out_size;

    for (int f = 0; f < C_out; f += 1)
    {
        for (int w = 0; w < W_in; w += 1)
        {
            for (int h = 0; h < H_in; h += 1)
            {
                double *o_curr = curr_out + mat_size * f + row_major(h, w, W_in);
                for (int c = 0; c < C_in; c += 1)
                {
                    double *f_curr = F_1D + f * C_in + c;
                    double *inp_curr = curr_img + mat_size * c + row_major(h, w, W_in);
                    *o_curr += (*f_curr) * (*inp_curr);
                }
            }
        }
    }
}

void print_tensor(double *X, int size, const char *name)
{
    fprintf(stderr, "%s\n", name);
    for (int i = 0; i < size; i += 1)
    {
        fprintf(stderr, "%f ", X[i]);
    }
    fprintf(stderr, "\n");
}

void init_conv(int bbpw, int fbpw, int wbpw, int hbpw, int cbpw, int bbdw, int cbdw, int fdw, int hbdw, int wbdw, int hfdw, int wfbdw) {
}

void dws_conv(double *X, double *F_DW, double *F_1D, double *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int C_out, int stride_h, int stride_w, double* depthwise_output)
{
    particle_t* X_gpu;
    particle_t* F_DW_gpu;
    particle_t* F_1D_gpu;
    particle_t* O_gpu;
    particle_t* depthwise_output_gpu;

    hipMalloc((void**) &X_gpu, B * C_in * W_in * H_in * sizeof(double));
    hipMalloc((void**) &F_DW_gpu, N_dw * C_in * H_f * W_f * sizeof(double));
    hipMalloc((void**) &F_1D_gpu, N_1d * C_in * N_dw * sizeof(double));
    hipMalloc((void**) &O_gpu, B * C_out * W_out * H_out * sizeof(double));
    hipMalloc((void**) &depthwise_output_gpu, B * W_out * H_out * C_in * N_dw * sizeof(double));

    hipMemcpy(X_gpu, X, B * C_in * W_in * H_in * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(F_DW_gpu, F_DW, N_dw * C_in * H_f * W_f * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(F_1D_gpu, F_1D, N_1d * C_in * N_dw * sizeof(double), hipMemcpyHostToDevice);

    dim3 dimGrid(B, C_in);
    dim3 dimBlock(NUM_THREADS, NUM_THREADS);
    dw_conv<<<dimGrid, dimBlock>>>(X_gpu, F_DW_gpu, depthwise_output_gpu, B, H_in, W_in, C_in, H_f, W_f, N_dw, H_out, W_out, stride_h, stride_w);
    pw_conv<<<B, NUM_THREADS>>>(depthwise_output_gpu, F_1D_gpu, O_gpu, B, H_out, W_out, C_in * N_dw, C_out);

    hipMemcpy(O, O_gpu, B * C_out * W_out * H_out * sizeof(double), hipMemcpyDeviceToHost);
}
