#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "dws-gpu.h"

#define NUM_THREADS 32
#define min(a, b) (((a) < (b)) ? (a) : (b))
#define row_major(i, j, num_rows) ((i) * (num_rows) + (j))

__device__ int FILTER_BLOCK_PW;
__device__ int CHANNEL_BLOCK_PW;

__device__ int CHANNEL_BLOCK_DW;
__device__ int HEIGHT_FILTER_BLOCK_DW;
__device__ int WIDTH_FILTER_BLOCK_DW;

__device__ void dw_conv_gpu_blocked(float *X, float *F_DW, float *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int stride_h, int stride_w, int b, int c_, int f, int w, int h, int w_f_, int h_f_) {

    // Pre-computations
    int mat_size = W_in * H_in;
    int f_size = W_f * H_f;
    int img_size = mat_size * C_in;

    int temp_out_img_size = W_out * H_out;
    int temp_out_size = temp_out_img_size * C_in;

    // PTRS TO IMG IN BATCH
    float *curr_img = X + b * img_size;
    float *curr_out = O + b * temp_out_size;

    int C_b = min(CHANNEL_BLOCK_DW, C_in - c_);
    int W_f_b = min(WIDTH_FILTER_BLOCK_DW, W_f - w_f_);
    int H_f_b = min(HEIGHT_FILTER_BLOCK_DW, H_f - h_f_);

    // Filters are 2D
    for (int c = 0; c < C_b; c += 1)
    {
        float *curr_channel = curr_img + mat_size * (c + c_);
        for (int w_f = 0; w_f < W_f_b; w_f += 1)
        {
            for (int h_f = 0; h_f < H_f_b; h_f += 1)
            {
                // PTR TO CURRENT POSITION IN FILTER
                float *f_curr = F_DW + f_size * (c + c_) + row_major((h_f + h_f_), (w_f + w_f_), W_f);

                // PTR TO INPUT POSITION
                int h_curr = (h_f + h_f_) + stride_h * h;
                int w_curr = (w_f + w_f_) + stride_w * w;
                float *curr_inp = curr_channel + row_major(h_curr, w_curr, W_in);

                // PTR TO INPUT POSITION
                float *curr_out_xy = curr_out + temp_out_img_size * (c + c_) + row_major(h, w, W_out);

                // CONVOLVE
                *curr_out_xy = *curr_out_xy + *f_curr * *curr_inp;
            }
        }
    }
}

__global__ void dw_conv_gpu(float *X, float *F_DW, float *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int stride_h, int stride_w) {
	
	// Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x;
    int w = threadIdx.y + blockIdx.y * blockDim.y;
    int h = threadIdx.z + blockIdx.z * blockDim.z;

    if (b >= B || h >= H_out || w >= W_out) {
        return;
    }

    // Filters are 2D
    for (int c = 0; c < C_in; c += CHANNEL_BLOCK_DW)
    {
        for (int w_f = 0; w_f < W_f; w_f += WIDTH_FILTER_BLOCK_DW)
        {
            for (int h_f = 0; h_f < H_f; h_f += HEIGHT_FILTER_BLOCK_DW)
            {
                dw_conv_gpu_blocked(X, F_DW, O, B, H_in, W_in, C_in, H_f, W_f, N_dw, H_out, W_out, 
                                    stride_h, stride_w, b, c, 0, w, h, w_f, h_f);
            }
        }
    }
}

__device__ void pw_conv_gpu_blocked(float *X, float *F_1D, float *O, int B, int H_in, int W_in, int C_in, int C_out, int b, int f_, int w, int h, int c_)
{
    int mat_size = W_in * H_in;
    int img_size = mat_size * C_in;
    int out_size = mat_size * C_out;

    float *curr_img = X + b * img_size;
    float *curr_out = O + b * out_size;

    int F_b = min(FILTER_BLOCK_PW, C_out - f_);
    int C_b = min(CHANNEL_BLOCK_PW, C_in - c_);

    for (int f = 0; f < F_b; f += 1)
    {
        float *o_curr = curr_out + mat_size * (f + f_) + row_major(h, w, W_in);
        for (int c = 0; c < C_b; c += 1)
        {
            float *f_curr = F_1D + (f + f_) * C_in + (c + c_);
            float *inp_curr = curr_img + mat_size * (c + c_) + row_major(h, w, W_in);
            *o_curr += (*f_curr) * (*inp_curr);
        }
    }
}

__global__ void pw_conv_gpu(float *X, float *F_1D, float *O, int B, int H_in, int W_in, int C_in, int C_out)
{
    // Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x;
    int w = threadIdx.y + blockIdx.y * blockDim.y;
    int h = threadIdx.z + blockIdx.z * blockDim.z;

    if (b >= B || w >= W_in || h >= H_in) {
        return;
    }

    for (int f = 0; f < C_out; f += FILTER_BLOCK_PW)
    {
        for (int c = 0; c < C_in; c += CHANNEL_BLOCK_PW)
        {
            pw_conv_gpu_blocked(X, F_1D, O, B, H_in, W_in, C_in, C_out, b, f, w, h, c);
        }
    }
}

__global__ void init_conv_gpu(int bbpw, int fbpw, int wbpw, int hbpw, int cbpw, int bbdw, int cbdw, int fdw, int hbdw, int wbdw, int hfdw, int wfbdw) {
    FILTER_BLOCK_PW = fbpw;
    CHANNEL_BLOCK_PW = cbpw;
    
    CHANNEL_BLOCK_DW = cbdw;
    HEIGHT_FILTER_BLOCK_DW = hfdw;
    WIDTH_FILTER_BLOCK_DW = wfbdw;
}

void init_conv(int bbpw, int fbpw, int wbpw, int hbpw, int cbpw, int bbdw, int cbdw, int fdw, int hbdw, int wbdw, int hfdw, int wfbdw) {
    init_conv_gpu<<<1, 1>>>(bbpw, fbpw, wbpw, hbpw, cbpw, bbdw, cbdw, fdw, hbdw, wbdw, hfdw, wfbdw);
}

void dws_conv(float *X, float *F_DW, float *F_1D, float *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int C_out, int stride_h, int stride_w, float* depthwise_output)
{
    dim3 dw_gridDim(B, W_out, H_out);
    dim3 dw_blockDim(NUM_THREADS, NUM_THREADS);
    dw_conv_gpu<<<dw_gridDim, dw_blockDim>>>(X, F_DW, depthwise_output, B, H_in, W_in, C_in, H_f, W_f, N_dw, H_out, W_out, stride_h, stride_w);
    
    dim3 pw_gridDim(B, W_out, H_out);
    dim3 pw_blockDim(NUM_THREADS, NUM_THREADS);
    pw_conv_gpu<<<pw_gridDim, pw_blockDim>>>(depthwise_output, F_1D, O, B, H_out, W_out, C_in * N_dw, C_out);
}
