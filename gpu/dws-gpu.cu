#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "dws-gpu.h"

#define NUM_THREADS 16
#define min(a, b) (((a) < (b)) ? (a) : (b))
#define row_major(i, j, num_rows) ((i) * (num_rows) + (j))

__global__ void dw_conv_gpu(float *X, float *F_DW, float *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int stride_h, int stride_w) {
	
	// Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x;
    int w = threadIdx.y + blockIdx.y * blockDim.y;
    int h = threadIdx.z + blockIdx.z * blockDim.z;

    if (b >= B || h >= H_out || w >= W_out) {
        return;
    }

    // Pre-computations
    int mat_size = W_in * H_in;
    int f_size = W_f * H_f;
    int img_size = mat_size * C_in;

    int temp_out_img_size = W_out * H_out;
    int temp_out_size = temp_out_img_size * N_dw * C_in;

    // PTRS TO IMG IN BATCH
    float *curr_img = X + b * img_size;
    float *curr_out = O + b * temp_out_size;

    // Filters are 2D
    for (int c = 0; c < C_in; c += 1)
    {
        float *curr_channel = curr_img + mat_size * c;
        // MICROKERNEL - tile if needed.
        for (int w_f = 0; w_f < W_f; w_f += 1)
        {
            for (int h_f = 0; h_f < H_f; h_f += 1)
            {
                // PTR TO CURRENT POSITION IN FILTER
                float *f_curr = F_DW + f_size * c + row_major(h_f, w_f, W_f);

                // PTR TO INPUT POSITION
                int h_curr = h_f + stride_h * h;
                int w_curr = w_f + stride_w * w;
                float *curr_inp = curr_channel + row_major(h_curr, w_curr, W_in);

                // PTR TO INPUT POSITION
                float *curr_out_xy = curr_out + temp_out_img_size * c + row_major(h, w, W_out);

                // CONVOLVE
                *curr_out_xy = *curr_out_xy + *f_curr * *curr_inp;
            }
        }
    }
}

__global__ void pw_conv_gpu(float *X, float *F_1D, float *O, int B, int H_in, int W_in, int C_in, int C_out)
{
    // Compute batch and channel for this thread
    int b = threadIdx.x + blockIdx.x * blockDim.x;
    int w = threadIdx.y + blockIdx.y * blockDim.y;
    int h = threadIdx.z + blockIdx.z * blockDim.z;

    if (b >= B || w >= W_in || h >= H_in) {
        return;
    }

    int mat_size = W_in * H_in;
    int img_size = mat_size * C_in;
    int out_size = mat_size * C_out;

    float *curr_img = X + b * img_size;
    float *curr_out = O + b * out_size;

    for (int f = 0; f < C_out; f += 1)
    {
        float *o_curr = curr_out + mat_size * f + row_major(h, w, W_in);
        for (int c = 0; c < C_in; c += 1)
        {
            float *f_curr = F_1D + f * C_in + c;
            float *inp_curr = curr_img + mat_size * c + row_major(h, w, W_in);
            *o_curr += (*f_curr) * (*inp_curr);
        }
    }
}

void print_tensor(float *X, int size, const char *name)
{
    fprintf(stderr, "%s\n", name);
    for (int i = 0; i < size; i += 1)
    {
        fprintf(stderr, "%f ", X[i]);
    }
    fprintf(stderr, "\n");
}

void init_conv(int bbpw, int fbpw, int wbpw, int hbpw, int cbpw, int bbdw, int cbdw, int fdw, int hbdw, int wbdw, int hfdw, int wfbdw) {
}

void dws_conv(float *X, float *F_DW, float *F_1D, float *O, int B, int H_in, int W_in, int C_in, int H_f, int W_f, int N_dw, int H_out, int W_out, int C_out, int stride_h, int stride_w, float* depthwise_output)
{
    dim3 dw_gridDim(B, W_out, H_out);
    dim3 dw_blockDim(NUM_THREADS, NUM_THREADS);
    dw_conv_gpu<<<dw_gridDim, dw_blockDim>>>(X, F_DW, depthwise_output, B, H_in, W_in, C_in, H_f, W_f, N_dw, H_out, W_out, stride_h, stride_w);
    
    dim3 pw_gridDim(B, W_out, H_out);
    dim3 pw_blockDim(NUM_THREADS, NUM_THREADS);
    pw_conv_gpu<<<pw_gridDim, pw_blockDim>>>(depthwise_output, F_1D, O, B, H_out, W_out, C_in * N_dw, C_out);
}
